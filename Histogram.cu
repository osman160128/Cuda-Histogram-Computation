#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#define NUM_BINS 256  // Number of histogram bins (e.g., for grayscale image, 256 bins for pixel intensities)

// CUDA kernel to compute histogram in parallel
__global__ void compute_histogram(int *data, int *histogram, int data_size) {
    __shared__ int shared_histogram[NUM_BINS];

    // Initialize shared memory
    if (threadIdx.x < NUM_BINS) {
        shared_histogram[threadIdx.x] = 0;
    }
    __syncthreads();

    // Calculate the start and end indices for this thread
    int start_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Traverse the data in chunks and update histogram bins
    for (int i = start_idx; i < data_size; i += stride) {
        int value = data[i];
        atomicAdd(&shared_histogram[value], 1);
    }

    __syncthreads();

    // Update global histogram from shared memory
    if (threadIdx.x < NUM_BINS) {
        atomicAdd(&histogram[threadIdx.x], shared_histogram[threadIdx.x]);
    }
}

void run_histogram_kernel(int *data, int *histogram, int data_size, int num_blocks, int threads_per_block) {
    int *d_data, *d_histogram;
    hipMalloc(&d_data, data_size * sizeof(int));
    hipMalloc(&d_histogram, NUM_BINS * sizeof(int));

    hipMemcpy(d_data, data, data_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_histogram, 0, NUM_BINS * sizeof(int));  // Initialize histogram to 0

    // Launch the kernel
    compute_histogram<<<num_blocks, threads_per_block>>>(d_data, d_histogram, data_size);

    // Copy result back to host
    hipMemcpy(histogram, d_histogram, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_histogram);
}

int main() {
    // Example: generate random data
    int data_size = 100000000;  // Size of the data (e.g., pixels in an image)
    std::vector<int> data(data_size);
    for (int i = 0; i < data_size; i++) {
        data[i] = rand() % NUM_BINS;  // Random values between 0 and NUM_BINS-1
    }

    int *histogram = new int[NUM_BINS]{0};  // Histogram initialization
    int num_blocks = 128;  // Number of blocks
    int threads_per_block = 256;  // Number of threads per block

    // Run the histogram computation on the GPU
    run_histogram_kernel(data.data(), histogram, data_size, num_blocks, threads_per_block);

    // Output the histogram
    for (int i = 0; i < NUM_BINS; i++) {
        std::cout << "Bin " << i << ": " << histogram[i];
    }

    delete[] histogram;

    std::cout<<std::endl;
    return 0;
}
